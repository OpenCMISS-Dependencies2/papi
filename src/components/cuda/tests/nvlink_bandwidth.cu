#include "hip/hip_runtime.h"
/* 
 * Copyright 2015-2016 NVIDIA Corporation. All rights reserved.
 *
 * Sample to demonstrate use of NVlink CUPTI APIs
 * 
 * This version is significantly changed to use PAPI and the CUDA component to
 * handle access and reporting. As of 10/05/2018, I have deleted all CUPTI_ONLY
 * references, for clarity. The file nvlink_bandwidth_cupti_only.cu contains
 * the cupti-only code.  I also deleted the #if PAPI; there is no option
 * without PAPI.  Also, before my changes, the makefile did not even have a
 * build option that set CUPTI_ONLY for this file.
 *
 * -TonyC. 
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include "papi.h"

// THIS MACRO EXITS if the papi call does not return PAPI_OK. Do not use for routines that
// return anything else; e.g. PAPI_num_components, PAPI_get_component_info, PAPI_library_init.
#define CALL_PAPI_OK(papi_routine)                                                        \
    do {                                                                                  \
        int _papiret = papi_routine;                                                      \
        if (_papiret != PAPI_OK) {                                                        \
            fprintf(stderr, "%s:%d: PAPI Error: function %s failed with ret=%d [%s].\n",  \
                    __FILE__, __LINE__, #papi_routine, _papiret, PAPI_strerror(_papiret));\
            exit(-1);                                                                     \
        }                                                                                 \
    } while (0);


#define CUPTI_CALL(call)                                                \
    do {                                                                \
        CUptiResult _status = call;                                     \
        if (_status != CUPTI_SUCCESS) {                                 \
            const char *errstr;                                         \
            cuptiGetResultString(_status, &errstr);                     \
            fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
                    __FILE__, __LINE__, #call, errstr);                 \
            exit(-1);                                                   \
        }                                                               \
    } while (0);  

#define DRIVER_API_CALL(apiFuncCall)                                    \
    do {                                                                \
        hipError_t _status = apiFuncCall;                                 \
        if (_status != hipSuccess) {                                  \
            const char *errName=NULL, *errStr=NULL;                     \
            hipError_t _e1 = hipDrvGetErrorName(_status, &errName);           \
            hipError_t _e2 = hipDrvGetErrorString(_status, &errStr);          \
            fprintf(stderr, "%s:%d: error: function %s failed with error %d [%s]='%s'.\n", \
                    __FILE__, __LINE__, #apiFuncCall, _status, errName, errStr);           \
            exit(-1);                                                   \
        }                                                               \
    } while (0);  

#define RUNTIME_API_CALL(apiFuncCall)                                   \
    do {                                                                \
        hipError_t _status = apiFuncCall;                              \
        if (_status != hipSuccess) {                                   \
            fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
                    __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status)); \
            exit(-1);                                                   \
        }                                                               \
    } while (0);  

#define MEMORY_ALLOCATION_CALL(var)                                     \
    do {                                                                \
        if (var == NULL) {                                              \
            fprintf(stderr, "%s:%d: Error: Memory Allocation Failed \n", \
                    __FILE__, __LINE__);                                \
            exit(-1);                                                   \
        }                                                               \
    } while (0);  


#define MAX_DEVICES    (32)
#define BLOCK_SIZE     (1024)
#define GRID_SIZE      (512)
#define BUF_SIZE       (32 * 1024)
#define ALIGN_SIZE     (8)
#define SUCCESS        (0)
#define NUM_METRIC     (2)
#define MAX_SIZE       (64*1024*1024)   // 64 MB

int Streams;                            // Number of physical copy engines to use; taken from Device Properties asyncEngineCount.
int cpuToGpu = 0;
int gpuToGpu = 0;


//-----------------------------------------------------------------------------
// This is the GPU routine to move a block from dst (on one GPU) to src (on
// another GPU. 
//-----------------------------------------------------------------------------
extern "C" __global__ void test_nvlink_bandwidth(float *src, float *dst)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = src[idx] * 2.0f;
} // end routine

#define DIM(x) (sizeof(x)/sizeof(*(x)))


//-----------------------------------------------------------------------------
// Return a text version with B, KB, MB, GB or TB. 
//-----------------------------------------------------------------------------
void calculateSize(char *result, uint64_t size)
{
    int i;

    const char *sizes[] = { "TB", "GB", "MB", "KB", "B" };
    uint64_t exbibytes = 1024ULL * 1024ULL * 1024ULL * 1024ULL;

    uint64_t multiplier = exbibytes;

    for(i = 0; (unsigned) i < DIM(sizes); i++, multiplier /= (uint64_t) 1024) {
        if(size < multiplier)
            continue;
        sprintf(result, "%.1f %s", (float) size / multiplier, sizes[i]);
        return;
    }
    strcpy(result, "0");
    return;
} // end routine


//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
void testCpuToGpu(CUpti_EventGroup * eventGroup, 
      hipDeviceptr_t * pDevBuffer, float **pHostBuffer, size_t bufferSize, 
      hipStream_t * cudaStreams, uint64_t * timeDuration, 
      int numEventGroup)
{
    int i;
    fprintf(stderr, "Streams = %d.\n", Streams); 
    // Unidirectional copy H2D (Host to Device).
    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    // Unidirectional copy D2H (Device to Host).
    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync(pHostBuffer[i], (void *) pDevBuffer[i], bufferSize, hipMemcpyDeviceToHost, cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    // Bidirectional copy
    for(i = 0; i < Streams; i += 2) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
        RUNTIME_API_CALL(hipMemcpyAsync(pHostBuffer[i + 1], (void *) pDevBuffer[i + 1], bufferSize, hipMemcpyDeviceToHost, cudaStreams[i + 1]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());
} // end routine.


//-----------------------------------------------------------------------------
// Copy buffers from the host to each device, in preparation for a transfer
// between devices.
//-----------------------------------------------------------------------------
void testGpuToGpu_part1(CUpti_EventGroup * eventGroup, 
      hipDeviceptr_t * pDevBuffer0, hipDeviceptr_t * pDevBuffer1, 
      float **pHostBuffer, size_t bufferSize, 
      hipStream_t * cudaStreams, uint64_t * timeDuration, 
      int numEventGroup)
{
    int i;

    RUNTIME_API_CALL(hipSetDevice(0));
    RUNTIME_API_CALL(hipDeviceEnablePeerAccess(1, 0));
    RUNTIME_API_CALL(hipSetDevice(1));
    RUNTIME_API_CALL(hipDeviceEnablePeerAccess(0, 0));

    // Unidirectional copy H2D
    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer0[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
    }

    RUNTIME_API_CALL(hipDeviceSynchronize());

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer1[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
    }

    RUNTIME_API_CALL(hipDeviceSynchronize());
} // end routine.


//-----------------------------------------------------------------------------
// Copy from device zero to device 1, then from device 1 to device 0.
//-----------------------------------------------------------------------------
void testGpuToGpu_part2(CUpti_EventGroup * eventGroup, 
      hipDeviceptr_t * pDevBuffer0, hipDeviceptr_t * pDevBuffer1, 
      float **pHostBuffer, size_t bufferSize, 
      hipStream_t * cudaStreams, uint64_t * timeDuration, 
      int numEventGroup)
{
    int i;

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer0[i], (void *) pDevBuffer1[i], bufferSize, hipMemcpyDeviceToDevice, cudaStreams[i]));
        //printf("Copy %zu stream %d to devBuffer0 from devBuffer1 \n", bufferSize, i);
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer1[i], (void *) pDevBuffer0[i], bufferSize, hipMemcpyDeviceToDevice, cudaStreams[i]));
        // printf("Copy %zu stream %d to devBuffer0 from devBuffer1 \n", bufferSize, i);
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    for(i = 0; i < Streams; i++) {
        test_nvlink_bandwidth <<< GRID_SIZE, BLOCK_SIZE >>> ((float *) pDevBuffer1[i], (float *) pDevBuffer0[i]);
        // printf("test_nvlink_bandwidth stream %d \n", i);
    }
} // end routine.


//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
static void printUsage()
{
    printf("usage: Demonstrate use of NVlink CUPTI APIs\n");
    printf("       -help           : display help message\n");
    printf("       --cpu-to-gpu    : Show results for data transfer between CPU and GPU \n");
    printf("       --gpu-to-gpu    : Show results for data transfer between two GPUs \n");
} // end routine.


//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
void parseCommandLineArgs(int argc, char *argv[])
{
    if(argc != 2) {
        printf("Invalid number of options\n");
        exit(0);
    }

    if(strcmp(argv[1], "--cpu-to-gpu") == 0) {
        cpuToGpu = 1;
    } else if(strcmp(argv[1], "--gpu-to-gpu") == 0) {
        gpuToGpu = 1;
    } else if((strcmp(argv[1], "--help") == 0) || 
              (strcmp(argv[1], "-help") == 0)  || 
              (strcmp(argv[1], "-h") == 0)) {
        printUsage();
        exit(0);
    } else {
        cpuToGpu = 1;
    }
} // end routine.


//-----------------------------------------------------------------------------
// Main program.
//-----------------------------------------------------------------------------
int main(int argc, char *argv[])
{
    int deviceCount = 0, i = 0, numEventGroup = 0, eventsRead=0;
    size_t bufferSize = 0, freeMemory = 0, totalMemory = 0;
    char str[64];

    hipDeviceptr_t *pDevBuffer0  = NULL;
    hipDeviceptr_t *pDevBuffer1  = NULL;
    float       **pHostBuffer = NULL;

    hipStream_t *cudaStreams = NULL;
    hipDeviceProp_t prop[MAX_DEVICES];
    uint64_t timeDuration;
    CUpti_EventGroup eventGroup[32];

    // This contains the original set of 'bandwidth' events to read.  This
    // program attempts to read them together within a single EventSet. PAPI
    // allows them all, and the component does not complain, but it always
    // returns 0 for the two "receive" events below. That is not the result
    // when we read each of these separately for the same program; which is the
    // approach taken in the program nvlink_all.cu. I suspect nvlink has a
    // conflict of some sort, but haven't tracked down the documentation to
    // prove that.  -Tony C.

    const char *TransmitBase[NUM_METRIC] = {
         "cuda:::metric:nvlink_total_data_transmitted"
        ,"cuda:::metric:nvlink_transmit_throughput"
    };
    
    const char *ReceiveBase[NUM_METRIC] = {
         "cuda:::metric:nvlink_total_data_received"
        ,"cuda:::metric:nvlink_receive_throughput"
    };
    // Parse command line arguments
    parseCommandLineArgs(argc, argv);

//  CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_NVLINK));
//  CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted));


    DRIVER_API_CALL(hipInit(0));
    RUNTIME_API_CALL(hipGetDeviceCount(&deviceCount));
    printf("There are %d devices.\n", deviceCount);

    if(deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        exit(-1);
    }

    Streams = 1;                                            // Always use at least ONE stream.
    for(i = 0; i < deviceCount; i++) {
        RUNTIME_API_CALL(hipGetDeviceProperties(&prop[i], i));
        printf("CUDA Device %d Name: %s", i, prop[i].name);
        printf(", AsyncEngineCount=%i", prop[i].asyncEngineCount);
        printf(", MultiProcessors=%i", prop[i].multiProcessorCount);
        printf(", MaxThreadsPerMP=%i", prop[i].maxThreadsPerMultiProcessor);
        printf("\n");
        if (prop[i].asyncEngineCount > Streams) {           // If a new high,
            Streams = prop[i].asyncEngineCount;             // Always use the maximum.
        }
    }

    printf("Streams to use: %i (= max Copy Engines).\n", Streams);

    // allocate space
    pDevBuffer0 = (hipDeviceptr_t*)    calloc(Streams, sizeof(hipDeviceptr_t));
    pDevBuffer1 = (hipDeviceptr_t*)    calloc(Streams, sizeof(hipDeviceptr_t));
    pHostBuffer = (float **)        calloc(Streams, sizeof(float*));
    cudaStreams = (hipStream_t*)   calloc(Streams, sizeof(hipStream_t));

    // Set memcpy size based on available device memory
    RUNTIME_API_CALL(hipMemGetInfo(&freeMemory, &totalMemory));
    printf("Total Device Memory available : ");
    calculateSize(str, (uint64_t) totalMemory);
    printf("%s\n", str);

    bufferSize = MAX_SIZE < (freeMemory / 4) ? MAX_SIZE : (freeMemory / 4);
    bufferSize = bufferSize/2;
    printf("Memcpy size is set to %llu B (%llu MB)\n", (unsigned long long) bufferSize, (unsigned long long) bufferSize / (1024 * 1024));

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipStreamCreate(&cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    // Nvlink-topology Records are generated even before hipMemcpy API is called.
    CUPTI_CALL(cuptiActivityFlushAll(0));

    fprintf(stderr, "Setup PAPI counters internally (PAPI)\n");
    int EventSet = PAPI_NULL;
    long long values[MAX_DEVICES * NUM_METRIC];
    char *EventName[MAX_DEVICES * NUM_METRIC];
    int eventCount;
    int retval, ee;
    int k, cid=-1;

    /* PAPI Initialization */
    retval = PAPI_library_init(PAPI_VER_CURRENT);
    if(retval != PAPI_VER_CURRENT) {
        fprintf(stderr, "PAPI_library_init failed, ret=%i [%s]\n", 
            retval, PAPI_strerror(retval));
        exit(-1);
    }

    fprintf(stderr, "PAPI version: %d.%d.%d\n", 
        PAPI_VERSION_MAJOR(PAPI_VERSION), 
        PAPI_VERSION_MINOR(PAPI_VERSION), 
        PAPI_VERSION_REVISION(PAPI_VERSION));

    // Find cuda component index.
    k = PAPI_num_components();                                          // get number of components.
    for (i=0; i<k && cid<0; i++) {                                      // while not found,
        PAPI_component_info_t *aComponent = 
            (PAPI_component_info_t*) PAPI_get_component_info(i);        // get the component info.     
        if (aComponent == NULL) {                                       // if we failed,
            fprintf(stderr,  "PAPI_get_component_info(%i) failed, "
                "returned NULL. %i components reported.\n", i,k);
            exit(-1);    
        }

       if (strcmp("cuda", aComponent->name) == 0) cid=i;                // If we found our match, record it.
    } // end search components.

    if (cid < 0) {                                                      // if no PCP component found,
        fprintf(stderr, "Failed to find pcp component among %i "
            "reported components.\n", k);
        exit(-1); 
    }

    fprintf(stderr, "Found CUDA Component at id %d\n",cid);

    CALL_PAPI_OK(PAPI_create_eventset(&EventSet)); 
    CALL_PAPI_OK(PAPI_assign_eventset_component(EventSet, cid)); 

    // ===== Allocate Memory =====================================

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMalloc((void **) &pDevBuffer0[i], bufferSize));

        pHostBuffer[i] = (float *) malloc(bufferSize);
        MEMORY_ALLOCATION_CALL(pHostBuffer[i]);
    }
    
    // Add events at a GPU specific level ... eg cuda:::metric:nvlink_total_data_transmitted:device=0
    char tmpEventName[1024];
    eventCount = 0;
    for(i = 0; i < deviceCount; i++) {                                  // Profile all devices.
        fprintf(stderr, "Set device to %d\n", i);
        for(ee = 0; ee < NUM_METRIC; ee++) {
            snprintf(tmpEventName, 1024, "%s:device=%d\0", TransmitBase[ee], i);
            retval = PAPI_add_named_event(EventSet, tmpEventName);      // Don't want to fail program if name not found...
            if(retval == PAPI_OK) {
                fprintf(stderr, "Added event %s to GPU %d.\n", tmpEventName, i);
                EventName[eventCount] = strdup(tmpEventName);
                eventCount++;
            } else {
                fprintf(stderr, "Failed to add event %s to GPU %i; ret=%d [%s].\n", tmpEventName, i, retval, PAPI_strerror(retval));
            }
        }
    }

    if (eventCount > 0) {                                              // If we have events...
        for(i = 0; i < eventCount; i++) values[i] = -1;                // init.

        if(cpuToGpu) {
            CALL_PAPI_OK(PAPI_start(EventSet));                             // Start event counters.
            testCpuToGpu(eventGroup, pDevBuffer0, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            CALL_PAPI_OK(PAPI_stop(EventSet, values));                      // Stop and read values.
        } else if(gpuToGpu) {
            RUNTIME_API_CALL(hipSetDevice(1));
            for(i = 0; i < Streams; i++) 
                RUNTIME_API_CALL(hipMalloc((void **) &pDevBuffer1[i], bufferSize));

            //  Prepare the copy, load up buffers on each device from the host.
            testGpuToGpu_part1(eventGroup, pDevBuffer0, pDevBuffer1, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            CALL_PAPI_OK(PAPI_start(EventSet));                             // Start event counters.
            // Copy from device 0->1, then device 1->0.
            testGpuToGpu_part2(eventGroup, pDevBuffer0, pDevBuffer1, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            CALL_PAPI_OK(PAPI_stop(EventSet, values));                      // Stop and read values.
        }

        // report each event counted.
        for(i = 0; i < eventCount; i++) {
            if (values[i] >= 0) {                                           // If not still -1,
                eventsRead++;                                               // .. count and report.
                calculateSize(str, (uint64_t) values[i] );
                printf("PAPI %64s: %s \n", EventName[i], str);
            }
        }
    }

    CALL_PAPI_OK(PAPI_cleanup_eventset(EventSet));                      // Delete all events in set.

    // Now start over, with receive events. Pretty much the same as above.
    eventCount = 0;

    // Add events at a GPU specific level ... eg cuda:::metric:nvlink_total_data_received:device=0
    for(i = 0; i < deviceCount; i++) {                                  // Profile all devices.
        fprintf(stderr, "Set device to %d\n", i);
        for(ee = 0; ee < NUM_METRIC; ee++) {
            snprintf(tmpEventName, 1024, "%s:device=%d\0", ReceiveBase[ee], i);
            retval = PAPI_add_named_event(EventSet, tmpEventName);      // Don't want to fail program if name not found...
            if(retval == PAPI_OK) {
                fprintf(stderr, "Added event %s to GPU %d.\n", tmpEventName, i);
                EventName[eventCount] = strdup(tmpEventName);
                eventCount++;
            } else {
                fprintf(stderr, "Failed to add event %s to GPU %i; ret=%d [%s].\n", tmpEventName, i, retval, PAPI_strerror(retval));
            }
        }
    }

    if (eventCount > 0) {                                               // If we have events...
        for(i = 0; i < eventCount; i++) values[i] = -1;                     // init.

        if(cpuToGpu) {
            CALL_PAPI_OK(PAPI_start(EventSet));                             // Start event counters.
            testCpuToGpu(eventGroup, pDevBuffer0, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            CALL_PAPI_OK(PAPI_stop(EventSet, values));                      // Stop and read values.
        } else if(gpuToGpu) {
            RUNTIME_API_CALL(hipSetDevice(1));
            for(i = 0; i < Streams; i++) 
                RUNTIME_API_CALL(hipMalloc((void **) &pDevBuffer1[i], bufferSize));

            //  Prepare the copy, load up buffers on each device from the host.
            testGpuToGpu_part1(eventGroup, pDevBuffer0, pDevBuffer1, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            CALL_PAPI_OK(PAPI_start(EventSet));                             // Start event counters.
            // Copy from device 0->1, then device 1->0.
            testGpuToGpu_part2(eventGroup, pDevBuffer0, pDevBuffer1, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            CALL_PAPI_OK(PAPI_stop(EventSet, values));                      // Stop and read values.
        }

        // report each event counted.
        for(i = 0; i < eventCount; i++) {
            if (values[i] >= 0) {                                           // If not still -1,
                eventsRead++;                                               // .. count and report.
                calculateSize(str, (uint64_t) values[i] );
                printf("PAPI %64s: %s \n", EventName[i], str);
            }
        }
    }

    if (eventCount < 1) {                                               // If we couldn't report anything,
        printf("None of our transmit or receive events were successfully added to the eventsets.\n"); 
    }

    CALL_PAPI_OK(PAPI_cleanup_eventset(EventSet));                      // Delete all events in set.
    // END OF receive event reporting.

    // Program cleanup.
    CALL_PAPI_OK(PAPI_destroy_eventset(&EventSet));                     // Release PAPI memory.
    PAPI_shutdown();                                                    // Has no return.
        
    if (eventsRead > 0) {                                               // If we succeeded with any, report. 
        printf("%i bandwidth events successfully reported.\n", eventsRead);
        return(0);                                                      // exit OK.
    }

    printf("Failed to read any bandwidth events.\n");                   // report a failure.
        
    return (-1);                                                        // Exit with error.
} // end MAIN.
